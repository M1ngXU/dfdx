#include "hip/hip_runtime.h"
#include "cuda_utils.cuh"

template<typename T>
__device__ void from_sparse_fwd(
    const size_t numel,
    const T * values,
    const size_t *values_info,
    const size_t *indeces,
    const size_t *indeces_info,
    T* output,
    const size_t *output_info,
    const size_t num_dims
) {
    unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;

    if (i >= numel) {
        return;
    }

    const size_t *values_dims = values_info;
    const size_t *values_strides = values_info + 1;
    const size_t *indeces_dims = indeces_info;
    const size_t *indeces_strides = indeces_info + 2;
    const size_t *output_dims = output_info;
    const size_t *output_strides = output_info + num_dims;

    unsigned int index = 0;
    for (unsigned int d = num_dims - 1; d < num_dims; d--) {
        index += get_strided_index(i * num_dims + d, 2, indeces_dims, indeces_strides) * output_dims[d];
    }
    float value = values[get_strided_index(i, 1, values_dims, values_strides)];
    output[get_strided_index(index, num_dims, output_dims, output_strides)] = value;
}

template<typename T>
__device__ void from_sparse_bwd(
    const size_t numel,
    T * values,
    const size_t *values_info,
    const size_t *indeces,
    const size_t *indeces_info,
    const T* output,
    const size_t *output_info,
    const size_t num_dims
) {
    unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;

    if (i >= numel) {
        return;
    }

    const size_t *values_dims = values_info;
    const size_t *values_strides = values_info + 1;
    const size_t *indeces_dims = indeces_info;
    const size_t *indeces_strides = indeces_info + 2;
    const size_t *output_dims = output_info;
    const size_t *output_strides = output_info + num_dims;

    unsigned int index = 0;
    for (unsigned int d = num_dims - 1; d < num_dims; d--) {
        index += get_strided_index(i * num_dims + d, 2, indeces_dims, indeces_strides) * output_dims[d];
    }
    float value = output[get_strided_index(index, num_dims, output_dims, output_strides)];
    values[get_strided_index(i, 1, values_dims, values_strides)] = value;
}

#define FROM_SPARSE(TYPENAME, FWD, BWD) \
extern "C" __global__ void FWD( \
    const size_t numel, \
    const TYPENAME * values, \
    const size_t *values_info, \
    const size_t *indeces, \
    const size_t *indeces_info, \
    TYPENAME* output, \
    const size_t *output_info, \
    const size_t num_dims \
) { \
    from_sparse_fwd(numel, values, values_info, indeces, indeces_info, output, output_info, num_dims); \
} \
extern "C" __global__ void BWD( \
    const size_t numel, \
    TYPENAME * values, \
    const size_t *values_info, \
    const size_t *indeces, \
    const size_t *indeces_info, \
    const TYPENAME* output, \
    const size_t *output_info, \
    const size_t num_dims \
) { \
    from_sparse_bwd(numel, values, values_info, indeces, indeces_info, output, output_info, num_dims); \
}

FROM_SPARSE(__half, from_sparse_fwd_f16, from_sparse_bwd_f16);
FROM_SPARSE(float, from_sparse_fwd_f32, from_sparse_bwd_f32);
FROM_SPARSE(double, from_sparse_fwd_f64, from_sparse_bwd_f64);